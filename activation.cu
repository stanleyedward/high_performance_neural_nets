#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "activation_runner.cuh"

#define M 1024  // Number of rows /OUTPUT
#define N 1024  // Number of columns /BATCH_SIZE

int main()
{
    int size = M * N;
    size_t bytes = size * sizeof(float);

    printf("Activation Function\n");
    printf("Matrix: %d x %d\n\n", M, N);

    float *h_input = (float *)malloc(bytes);
    float *h_act1 = (float *)malloc(bytes);
    float *h_act2 = (float *)malloc(bytes);
    float *h_cpu_out = (float *)malloc(bytes);

    // init input with random values between -10 to 10
    for (int i = 0; i < size; i++)
    {
        h_input[i] = (float)rand() / RAND_MAX * 20.0f - 10.0f;
    }

    float *d_input, *d_act1, *d_act2;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_act1, bytes);
    hipMalloc(&d_act2, bytes);

    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    //warmup
    // run_kernel_softmax(1, M, N, d_input, d_act1);
    run_kernel_relu(1, M, N, d_input, d_act1);
    
    hipEventRecord(start);
    run_kernel_softmax(1, M, N, d_input, d_act1);
    // run_kernel_relu(1, M, N, d_input, d_act1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("act1 execution time: %.4f ms\n", milliseconds);
    float gflops_act1 = (8.0f * M * N) / (milliseconds * 1e6);
    printf("act1 GFLOPS: %.4f\n", gflops_act1);
    printf("\n");

    hipEventRecord(start);
    run_kernel_softmax(2, M, N, d_input, d_act2);
    // run_kernel_relu(2, M, N, d_input, d_act2);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("act2 execution time: %.4f ms\n", milliseconds);

    hipGetLastError();
    float gflops_act2 = (7.0f * M * N) / (milliseconds * 1e6);
    printf("act2 GFLOPS: %.4f\n", gflops_act2);
    hipMemcpy(h_act1, d_act1, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_act2, d_act2, bytes, hipMemcpyDeviceToHost);
    printf("\n");

    run_kernel_softmax(0, M, N, h_input, h_cpu_out);
    // run_kernel_relu(0, M, N, h_input, h_cpu_out);
    verify_results_activation(M, N, h_act1, h_act2, h_cpu_out);

    hipFree(d_input);
    hipFree(d_act1);
    hipFree(d_act2);
    free(h_input);
    free(h_act1);
    free(h_act2);
    free(h_cpu_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
