#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024
#define THREADS_PER_BLOCK 1024

__global__ void act1(float *input, float *output)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N)
    {
        output[row * N + col] = 1.0f / (1.0f + expf(-input[row * N + col]));
    }
}

__global__ void act2(float *input, float *output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

int main()
{
    int size = N * N;
    size_t bytes = size * sizeof(float);

    float *h_input = (float *)malloc(bytes);
    float *h_sigmoid = (float *)malloc(bytes);
    float *h_relu = (float *)malloc(bytes);

    // init input with random values between -10 to 10
    for (int i = 0; i < size; i++)
    {
        h_input[i] = (float)rand() / RAND_MAX * 20.0f - 10.0f;
    }

    float *d_input, *d_sigmoid, *d_relu;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_sigmoid, bytes);
    hipMalloc(&d_relu, bytes);

    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    dim3 numThreadsPerBlock = dim3(32, 32, 1);
    dim3 numBlocks = dim3((N + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x, (N + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    hipEventRecord(start);
    act1<<<numBlocks, numThreadsPerBlock>>>(d_input, d_sigmoid);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Sigmoid execution time: %.4f ms\n", milliseconds);

    // Check for kernel errors

    int blockSize = THREADS_PER_BLOCK;
    int gridSize = (size + blockSize - 1) / blockSize;

    hipEventRecord(start);
    act2<<<gridSize, blockSize>>>(d_input, d_relu, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ReLU execution time: %.4f ms\n", milliseconds);

    hipGetLastError();

    // cudaMemcpy(h_sigmoid, d_sigmoid, bytes, cudaMemcpyDeviceToHost);
    // cudaMemcpy(h_relu, d_relu, bytes, cudaMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_sigmoid);
    hipFree(d_relu);
    free(h_input);
    free(h_sigmoid);
    free(h_relu);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}