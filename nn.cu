#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 1024
#define OUTPUT_SIZE 10
#define BATCH_SIZE 64
#define BLOCK_SIZE 16

typedef struct {
    float *weights1;
    float *weights2;
    float *weights3;  

    float *biases1;
    float *biases2;
    float *biases3;   

    float *grad_layer1;
    float *grad_layer2;
    float *grad_layer3;  
} NeuralNetwork;

typedef struct {
  float *x1;
  float *a1;

  float *x2;
  float *a2;
  
  float *x3;
  float *a3;

  float *losses;
} Outputs;

__global__ void linear_forward(int batch_size, int n, int out_w, float* input, float* weights, float* biases, float* output)
{
  const uint column = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    output[row*out_w+column] = biases[column];
    for(int i = 0; i < n; i++)
    {
      output[row*out_w+column] += weights[i*out_w + column] * input[row*n + i];
    }
  }
}

__global__ void linear_backward(int batch_size, int n, int out_w, float* weights, float* biases, float* d_l, float* out_d_l)
{
  const uint column = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    float dl = 0.f;
    for(int i = 0; i < n; i++)
    {
      float w = weights[i*out_w + column];
      dl += w*d_l[row*n + i];
    }
    out_d_l[row*out_w + column] = dl;
  }
}

__global__ void relu_forward(int w, int h, float* a, float* b)
{
  const uint column = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    float activation = a[row*w+column];
    b[row*w+column] =  activation > 0.f ? activation : 0.f;
  }
}

__global__ void relu_backwards(int w, int h, float* a, float* d_l, float* b)
{
  const uint column = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && column < w)
  {
    float activation = a[row*w+column];
    b[row*w+column] = activation > 0.f ? d_l[row*w+column] : 0.f;
  }
}

__global__ void softmax(int w, int h, float* a, float* b)
{
  const uint col = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    //subtract with maxval for numeric stability
    float maxval = a[row*w];
    for (int i = 1; i<w; i++)
    {
      maxval = max(maxval, a[row*w + i]);
    }
    float divisor = 0.f;
    for (int i = 0; i<w; i++)
    {
      divisor += exp(a[row*w + i] - maxval);
    }
    b[row*w + col] = exp(a[row*w + col]-maxval)/(divisor);
  }
}

__global__ void init_kaiming_normal(int W, int H, float* matrix){
    const uint row = blockDim.x * blockIdx.x + threadIdx.x;
    const uint col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < H && col < W){
    hiprandState state;
    hiprand_init(42, row*W+col, 0, &state);
    matrix[row*W + col] = hiprand_normal(&state)*sqrtf(2.f/H);
    }
}

__global__ void cross_entropy(int w, int h, float* preds, float* real, float* output)
{
  const uint idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < h)
  {
    float loss = 0.f;
    for (int i = 0; i<w; i++)
    {
      loss -= real[idx*w + i] * log(max(1e-6, preds[idx*w + i]));
    }
    output[idx] = loss;
  }
}

__global__ void cross_entropy_backwards(int w, int h, float* preds, float* real, float* output)
{
  const uint col = blockIdx.x*blockDim.x + threadIdx.x;
  const uint row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < h && col < w)
  {
    output[row*w + col] = preds[row*w + col] - real[row*w + col];
  }
}

void init_parameters(float* weights, float* biases, int W, int H)
{
// weights
  dim3 numBlocks = dim3(ceil(W/(float)BLOCK_SIZE), ceil(H/(float)BLOCK_SIZE), 1);
  dim3 numThreadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
  init_kaiming_normal<<<numBlocks, numThreadsPerBlock>>>(W, H, weights);

// biases
  numBlocks = dim3(ceil(H/(float)BLOCK_SIZE), 1, 1);
  numThreadsPerBlock = dim3(BLOCK_SIZE, 1, 1);
  init_kaiming_normal<<<numBlocks, numThreadsPerBlock>>>(1, H, biases);
}

void initialize_nn(NeuralNetwork* nn){
    hipMalloc(&nn->weights1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    hipMalloc(&nn->biases1, HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer1, HIDDEN_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights1, nn->biases1, HIDDEN_SIZE, INPUT_SIZE);

    hipMalloc(&nn->weights2, HIDDEN_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->biases2, HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer2, HIDDEN_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights1, nn->biases1, HIDDEN_SIZE, HIDDEN_SIZE);
    
    hipMalloc(&nn->weights3, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->biases3, OUTPUT_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer3, OUTPUT_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights3, nn->biases3, OUTPUT_SIZE, HIDDEN_SIZE);
}

void init_outputs(Outputs* op){
    hipMalloc((void**) &op->x1, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a1, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->x2, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a2, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->x3, OUTPUT_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a3, OUTPUT_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->losses, BATCH_SIZE*sizeof(float));
}

void forward(NeuralNetwork* nn, Outputs *op, float* input, float* labels){
    linear_forward<<<dim3(HIDDEN_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(BATCH_SIZE, INPUT_SIZE, HIDDEN_SIZE, input, nn->weights1, nn->biases1, op->x1);
    relu_forward<<<dim3(HIDDEN_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(HIDDEN_SIZE, BATCH_SIZE, op->x1, op->a1);
    linear_forward<<<dim3(HIDDEN_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(BATCH_SIZE, HIDDEN_SIZE, HIDDEN_SIZE, op->a1, nn->weights2, nn->biases2, op->x2);
    relu_forward<<<dim3(HIDDEN_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(HIDDEN_SIZE, BATCH_SIZE, op->x2, op->a2);
    linear_forward<<<dim3(OUTPUT_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(BATCH_SIZE, HIDDEN_SIZE, OUTPUT_SIZE, op->a2, nn->weights3, nn->biases3, op->x3);
    softmax<<<dim3(OUTPUT_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(OUTPUT_SIZE, BATCH_SIZE, op->x3, op->a3);
    cross_entropy<<<dim3(OUTPUT_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1, 1), dim3(BLOCK_SIZE, 1, 1)>>>(OUTPUT_SIZE, BATCH_SIZE, op->a3, labels, op->losses);
    hipDeviceSynchronize();
}

void backward(NeuralNetwork* nn, Outputs *op, float* labels){
    cross_entropy_backwards<<<dim3(OUTPUT_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(OUTPUT_SIZE, BATCH_SIZE, op->a3, labels, nn->grad_layer3);
    linear_backward<<<dim3(HIDDEN_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(BATCH_SIZE, OUTPUT_SIZE, HIDDEN_SIZE, nn->weights3, nn->biases3, nn->grad_layer3, nn->grad_layer2);
    relu_backwards<<<dim3(HIDDEN_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(HIDDEN_SIZE, BATCH_SIZE, op->a2, nn->grad_layer2, nn->grad_layer2);
    linear_backward<<<dim3(HIDDEN_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(BATCH_SIZE, HIDDEN_SIZE, INPUT_SIZE, nn->weights2, nn->biases2, nn->grad_layer2, nn->grad_layer1);
    relu_backwards<<<dim3(HIDDEN_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, BATCH_SIZE + BLOCK_SIZE - 1/(float)BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(HIDDEN_SIZE, BATCH_SIZE, op->a1, nn->grad_layer1, nn->grad_layer1);
    hipDeviceSynchronize();
}

void optimizer_step(){
  
}



int main(){
    NeuralNetwork nn;
    initialize_nn(&nn);

    Outputs op;
    init_outputs(&op);

return 0;
}