#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 1024
#define OUTPUT_SIZE 10
#define BATCH_SIZE 64
#define BLOCK_SIZE 16

typedef struct {
    float *weights1;
    float *weights2;
    float *weights3;  

    float *biases1;
    float *biases2;
    float *biases3;   

    float *grad_layer1;
    float *grad_layer2;
    float *grad_layer3;  
} NeuralNetwork;

typedef struct {
  float *x1;
  float *a1;

  float *x2;
  float *a2;
  
  float *x3;
  float *a3;

  float *losses;
} Outputs;

__global__ void matmut_add(int batch_size, int n, int out_w, float* input, float* weights, float* biases, float* output)
{
  int column = blockIdx.x*blockDim.x + threadIdx.x;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  if (row < batch_size && column < out_w)
  {
    output[row*out_w+column] = biases[column];
    for(int i = 0; i < n; i++)
    {
      output[row*out_w+column] += weights[i*out_w + column] * input[row*n + i];
    }
  }
}

__global__ void init_kaiming_normal(int W, int H, float* matrix){
    const uint row = blockDim.x * blockIdx.x + threadIdx.x;
    const uint col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < H && col < W){
    hiprandState state;
    hiprand_init(42, row*W+col, 0, &state);
    matrix[row*W + col] = hiprand_normal(&state)*sqrtf(2.f/H);
    }
}

void init_parameters(float* weights, float* biases, int W, int H)
{
// weights
  dim3 numBlocks = dim3(ceil(W/(float)BLOCK_SIZE), ceil(H/(float)BLOCK_SIZE), 1);
  dim3 numThreadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
  init_kaiming_normal<<<numBlocks, numThreadsPerBlock>>>(W, H, weights);

// biases
  numBlocks = dim3(ceil(H/(float)BLOCK_SIZE), 1, 1);
  numThreadsPerBlock = dim3(BLOCK_SIZE, 1, 1);
  init_kaiming_normal<<<numBlocks, numThreadsPerBlock>>>(1, H, biases);
}

void initialize_nn(NeuralNetwork* nn){
    hipMalloc(&nn->weights1, HIDDEN_SIZE * INPUT_SIZE * sizeof(float));
    hipMalloc(&nn->biases1, HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer1, HIDDEN_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights1, nn->biases1, HIDDEN_SIZE, INPUT_SIZE);

    hipMalloc(&nn->weights2, HIDDEN_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->biases2, HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer2, HIDDEN_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights1, nn->biases1, HIDDEN_SIZE, HIDDEN_SIZE);
    
    hipMalloc(&nn->weights3, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc(&nn->biases3, OUTPUT_SIZE * sizeof(float));
    hipMalloc(&nn->grad_layer3, OUTPUT_SIZE * BATCH_SIZE * sizeof(float));
    init_parameters(nn->weights3, nn->biases3, OUTPUT_SIZE, HIDDEN_SIZE);
}

void init_outputs(Outputs* op){
    hipMalloc((void**) &op->x1, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a1, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->x2, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a2, HIDDEN_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->x3, OUTPUT_SIZE*BATCH_SIZE*sizeof(float));
    hipMalloc((void**) &op->a3, OUTPUT_SIZE*BATCH_SIZE*sizeof(float));

    hipMalloc((void**) &op->losses, BATCH_SIZE*sizeof(float));
}

int main(){
    NeuralNetwork nn;
    initialize_nn(&nn);

    Outputs op;
    init_outputs(&op);

}